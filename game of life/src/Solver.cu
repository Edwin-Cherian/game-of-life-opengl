#include "hip/hip_runtime.h"
#include "Solver.h"
#include <iostream>
#include <chrono>
#include <vector>
Solver::Solver(float width, float height, int rows, int cols, int stride)
	: WINDOW_WIDTH(width), WINDOW_HEIGHT(height), ROWS(rows), COLS(cols), STRIDE(stride) {
	std::cout << WINDOW_HEIGHT << " ; " << WINDOW_WIDTH << std::endl;
};

void Solver::setCellState(float* positions, int row, int col, float state)
{
	positions[(row * COLS + col) * STRIDE +  2] = state;
	positions[(row * COLS + col) * STRIDE +  5] = state;
	positions[(row * COLS + col) * STRIDE +  8] = state;
	positions[(row * COLS + col) * STRIDE + 11] = state;
}

void Solver::resetCells(float* positions)
{
	for (int i = 0; i < ROWS * COLS; i++)
	{
		positions[i * STRIDE + 2] = 0.0f;
		positions[i * STRIDE + 5] = 0.0f;
		positions[i * STRIDE + 8] = 0.0f;
		positions[i * STRIDE + 11] = 0.0f;
	}
}



void Solver::updateCells(float* positions, float* positions_buffer)
{
	g_updateCells<<<256, 256>>>(positions, positions_buffer, STRIDE, ROWS, COLS);
	hipDeviceSynchronize();
	g_copyFromBuffer<<<256, 256>>>(positions, positions_buffer, STRIDE, ROWS, COLS);
	hipDeviceSynchronize();
}

int Solver::getNeighbours(float* positions, int idx)
{
	int count = -1 * positions[idx];
	for (int i = -1; i < 2; i++)
	{
		for (int j = -1; j < 2; j++)
		{
			if (idx + STRIDE * (i * COLS + j) >= 0 && idx + STRIDE * (i * COLS + j) <= ROWS * COLS * STRIDE)
				count += positions[idx + STRIDE * (i * COLS + j)];
		}
	}
	return count;
}

__global__
void g_updateCells(float* positions, float* positions_buffer, int stride, int rows, int cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_stride = blockDim.x * gridDim.x;
	for (int i = idx; i < rows * cols; i += thread_stride) {
		int neighbours = -1 * positions[i * stride + 2];
		for (int j = -1; j < 2; j++)
		{
			for (int k = -1; k < 2; k++)
			{
				if (stride * (i + j * cols + k) + 2 >= 0 && stride * (1 + j * cols + k) + 2 <= rows * cols * 12)
					neighbours += positions[stride * (i + j * cols + k) + 2];
			}
		}
		if ((positions[i * stride + 2] == 1.0f && neighbours == 2) || neighbours == 3) {
			positions_buffer[i * stride +  2] = 1.0f;
			positions_buffer[i * stride +  5] = 1.0f;
			positions_buffer[i * stride +  8] = 1.0f;
			positions_buffer[i * stride + 11] = 1.0f;
		}
		else {
			positions_buffer[i * stride +  2] = 0.0f;
			positions_buffer[i * stride +  5] = 0.0f;
			positions_buffer[i * stride +  8] = 0.0f;
			positions_buffer[i * stride + 11] = 0.0f;
		}
	}
}

__global__
void g_copyFromBuffer(float* positions, float* positions_buffer, int stride, int rows, int cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int thread_stride = blockDim.x * gridDim.x;
	for (int i = idx; i < rows * cols; i += thread_stride) {
		positions[i * stride +  2] = positions_buffer[i * stride +  2];
		positions[i * stride +  5] = positions_buffer[i * stride +  5];
		positions[i * stride +  8] = positions_buffer[i * stride +  8];
		positions[i * stride + 11] = positions_buffer[i * stride + 11];
	}
}



//void Solver::wallCollision(float* quadAttribIdx, float* velocityIdx, float p_size)
//{
//	if (*(quadAttribIdx + 0) < 0)
//	{
//		*(quadAttribIdx + 0) = 0;
//		*(velocityIdx + 0) *= -1;
//	}
//	if (*(quadAttribIdx + 0) > WINDOW_WIDTH - p_size)
//	{
//		*(quadAttribIdx + 0) = WINDOW_WIDTH - p_size;
//		*(velocityIdx + 0) *= -1;
//	}
//
//	if (*(quadAttribIdx + 1) < 0)
//	{
//		*(quadAttribIdx + 1) = 0;
//		*(velocityIdx + 1) *= -1;
//	}
//	if (*(quadAttribIdx + 1) > WINDOW_HEIGHT - p_size)
//	{
//		*(quadAttribIdx + 1) = WINDOW_HEIGHT - p_size;
//		*(velocityIdx + 1) *= -1;
//	}
//}
//
//__global__
//void mytest(std::vector<std::vector<std::vector<float*>>>* grid, float* positions, int p_count, int p_size, int stride)
//{
//	// empty kernel
//}
//
//void Solver::particleCollision2(float* positions, int p_count, int p_size, int stride, Grid* grid)
//{
//	for (int i = 0; i < p_count; i++)
//	{
//		std::vector<float*> nearby = grid->FindNear(&(positions[i * stride]));
//		int count = 0;
//		for (float* other : nearby)
//		{
//			// compute distance between current particle and nearby particles to test for collisions
//			if ((positions[i * stride + 0] - other[0]) * (positions[i * stride + 0] - other[0]) +
//				(positions[i * stride + 1] - other[1]) * (positions[i * stride + 1] - other[1]) <= p_size * p_size)
//			{
//				count++;
//			}
//		}
//		if (count > 1)
//		{
//			positions[i * stride + 4] = 1.0f;
//			positions[i * stride + 9] = 1.0f;
//			positions[i * stride + 14] = 1.0f;
//			positions[i * stride + 19] = 1.0f;
//		}
//		else
//		{
//			positions[i * stride + 4] = 0.0f;
//			positions[i * stride + 9] = 0.0f;
//			positions[i * stride + 14] = 0.0f;
//			positions[i * stride + 19] = 0.0f;
//		}
//	}
//}
//
//
//void Solver::particleCollision(float* quadAttribIdx, int p_count, int p_size, int stride, Grid* grid)
//{
//	//mytest <<<1, 1>>>();
//	std::vector<float*> nearby = grid->FindNear(quadAttribIdx);
//	int count = 0;
//	for (float* other : nearby)
//	{
//		// compute distance between current particle and nearby particles to test for collisions
//		if ((*(quadAttribIdx + 0) - *(other + 0)) * (*(quadAttribIdx + 0) - *(other + 0)) +
//			(*(quadAttribIdx + 1) - *(other + 1)) * (*(quadAttribIdx + 1) - *(other + 1)) <= p_size * p_size)
//		{
//			count++;
//		}
//	}
//	if (count > 1)
//	{
//		*(quadAttribIdx + 4) = 1.0f;
//		*(quadAttribIdx + 9) = 1.0f;
//		*(quadAttribIdx + 14) = 1.0f;
//		*(quadAttribIdx + 19) = 1.0f;
//	}
//	else
//	{
//		*(quadAttribIdx + 4) = 0.0f;
//		*(quadAttribIdx + 9) = 0.0f;
//		*(quadAttribIdx + 14) = 0.0f;
//		*(quadAttribIdx + 19) = 0.0f;
//	}
//}
//
//void Solver::updatePosition(float* quadAttribIdx, float* velocityIdx, float p_size, float dt)
//{
//	*(quadAttribIdx + 0) = *(quadAttribIdx + 0) + 1 * *(velocityIdx + 0);
//	*(quadAttribIdx + 1) = *(quadAttribIdx + 1) + 1 * *(velocityIdx + 1);
//	*(quadAttribIdx + 5) = *(quadAttribIdx + 0) + 1 * *(velocityIdx + 0) + p_size;
//	*(quadAttribIdx + 6) = *(quadAttribIdx + 1) + 1 * *(velocityIdx + 1);
//	*(quadAttribIdx + 10) = *(quadAttribIdx + 0) + 1 * *(velocityIdx + 0) + p_size;
//	*(quadAttribIdx + 11) = *(quadAttribIdx + 1) + 1 * *(velocityIdx + 1) + p_size;
//	*(quadAttribIdx + 15) = *(quadAttribIdx + 0) + 1 * *(velocityIdx + 0);
//	*(quadAttribIdx + 16) = *(quadAttribIdx + 1) + 1 * *(velocityIdx + 1) + p_size;
//}

